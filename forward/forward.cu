#include "hip/hip_runtime.h"
#include "forward.cuh"
#include <cstdio>

#include "included.cuh"
__device__
extern float log_sum_exp(float *probs, float neginf);

__device__
extern int64_t forward_prob_predecessors(HMM hmm, float *alpha_prev, int64_t instance, uint16_t state, float *probs);

template <unsigned int block_size>
__global__
void forward_init(HMM hmm, ObsSeqs seqs, float *alpha_src) {
  int64_t task_index = blockIdx.x;
  int64_t instance = task_index;
  uint8_t o = seqs.data[instance * seqs.maxlen];
  for (int64_t state = threadIdx.x; state < 1024; state += block_size) {
    alpha_src[instance * 1024 + state] = hmm.initial_prob[state] + hmm.output_prob[o * 64 + state % 64];
  }
}

template <unsigned int block_size>
__global__
void forward_step(HMM hmm, ObsSeqs seqs, float *alpha1, float *alpha2, int64_t t, float neginf) {
  int64_t task_index = blockIdx.x;
  int64_t instance = task_index;
  float *alpha_src, *alpha_dst;
  if (t & 1) {
    alpha_src = alpha1;
    alpha_dst = alpha2;
  } else {
    alpha_src = alpha2;
    alpha_dst = alpha1;
  }
  for (int64_t state = threadIdx.x; state < 1024; state += block_size) {
    int64_t idx = instance * 1024 + state;
    if (t < seqs.lens[instance]) {
      uint8_t o = seqs.data[instance * seqs.maxlen + t];
      float probs[5];
      int64_t pidx = forward_prob_predecessors(hmm, alpha_src, instance, state, probs);
      while (pidx < 5) {
        probs[pidx] = neginf;
        pidx = pidx + 1;
      }
      alpha_dst[idx] = log_sum_exp(probs, neginf) + hmm.output_prob[o * 64 + state % 64];
    } else if (t == seqs.lens[instance]) {
      alpha_dst[idx] = alpha_src[idx];
    }
  }
}

template <unsigned int block_size>
__global__
void forward_steps(HMM hmm, ObsSeqs seqs, float *alpha1, float *alpha2, float neginf) {
  int64_t task_index = blockIdx.x;
  int64_t instance = task_index;
  for (int64_t t = 1; t < seqs.maxlen; t++) {
    float *alpha_src, *alpha_dst;
    if (t & 1) {
      alpha_src = alpha1;
      alpha_dst = alpha2;
    } else {
      alpha_src = alpha2;
      alpha_dst = alpha1;
    }
    for (int64_t state = threadIdx.x; state < 1024; state += block_size) {
      int64_t idx = instance * 1024 + state;
      if (t < seqs.lens[instance]) {
        uint8_t o = seqs.data[instance * seqs.maxlen + t];
        float probs[5];
        int64_t pidx = forward_prob_predecessors(hmm, alpha_src, instance, state, probs);
        while (pidx < 5) {
          probs[pidx] = neginf;
          pidx = pidx + 1;
        }
        alpha_dst[idx] = log_sum_exp(probs, neginf) + hmm.output_prob[o * 64 + state % 64];
      } else if (t == seqs.lens[instance]) {
        alpha_dst[idx] = alpha_src[idx];
      }
    }
    // At the end of each iteration of the sequential outer loop, we need to
    // synchronize to ensure all results of the current iteration are available
    // at the start of the next iteration. The IR compiler has to insert this
    // to ensure the correct execution order.
    //
    // We synchronize as few threads as possible to reduce the overhead. In
    // particular, when we have at most 32 threads, we only need to sync a
    // single warp of threads (which should be faster than synchronizing
    // multiple threads).
    block_size > 32 ? __syncthreads() : __syncwarp();
  }
}

template <unsigned int block_size>
__global__
void forward_lse(HMM hmm, ObsSeqs seqs, float *alpha1, float *alpha2, float neginf, float *result) {
  int64_t task_index = blockIdx.x;
  float *alpha;
  if (seqs.maxlen & 1) {
    alpha = alpha1;
  } else {
    alpha = alpha2;
  }

  int64_t ofs = task_index * 1024;
  float maxp = neginf;
  for (int state = threadIdx.x; state < 1024; state += block_size) {
    maxp = max(maxp, alpha[ofs + state]);
  }

  // Warp-level synchronization.
  for (int i = 16; i > 0; i /= 2) {
    maxp = max(maxp, __shfl_xor_sync(0xFFFFFFFF, maxp, i));
  }

  // Block-level synchronization.
  __shared__ float temp[32];
  if (threadIdx.x % 32 == 0) {
    temp[threadIdx.x / 32] = maxp;
  }
  __syncthreads();
  if (threadIdx.x % 32 < block_size / 32) {
    maxp = temp[threadIdx.x % 32];
  } else {
    maxp = neginf;
  }

  for (int i = 16; i > 0; i /= 2) {
    if (block_size >= 64 * i) {
      maxp = max(maxp, __shfl_xor_sync(0xFFFFFFFF, maxp, i));
    }
  }

  // First thread writes result to index 0, which all threads read from after
  // synchronization to make them agree upon a single value.
  if (threadIdx.x == 0) {
    temp[0] = maxp;
  }
  __syncthreads();
  maxp = temp[0];

  // Compute the sum of exponentiated probabilities subtracted by the maximum
  // probability.
  float psum = 0.0;
  for (int state = threadIdx.x; state < 1024; state += block_size) {
    psum = psum + expf(alpha[ofs + state] - maxp);
  }

  // Warp-level synchronization.
  for (int i = 16; i > 0; i /= 2) {
    psum = psum + __shfl_xor_sync(0xFFFFFFFF, psum, i);
  }

  // Block-level synchronization.
  if (threadIdx.x % 32 == 0) {
    temp[threadIdx.x / 32] = psum;
  }
  __syncthreads();

  if (threadIdx.x % 32 < block_size / 32) {
    psum = temp[threadIdx.x % 32];
  } else {
    psum = 0.0;
  }

  for (int i = 16; i > 0; i /= 2) {
    if (block_size >= 64 * i) {
      psum = psum + __shfl_xor_sync(0xFFFFFFFF, psum, i);
    }
  }

  // First thread writes result to index 0, which all threads read from after
  // synchronization to make them agree upon a single value.
  if (threadIdx.x == 0) {
    temp[0] = psum;
  }
  __syncthreads();
  psum = temp[0];

  // Write the resulting value to global memory.
  if (threadIdx.x == 0) {
    result[task_index] = maxp + logf(psum);
  }
}

extern "C"
void forward(
    float hmm_gamma, float *hmm_trans1, float *hmm_trans2,
    float *hmm_output_prob, float *hmm_initial_prob, float hmm_synthetic_248,
    int64_t hmm_num_states, uint8_t *seqs_data, int64_t *seqs_lens,
    int64_t seqs_maxlen, int64_t seqs_num_instances, float *result,
    float *alpha1, float *alpha2) {
  HMM hmm;
  hmm.gamma = hmm_gamma;
  hmm.trans1 = hmm_trans1;
  hmm.trans2 = hmm_trans2;
  hmm.output_prob = hmm_output_prob;
  hmm.initial_prob = hmm_initial_prob;
  hmm.synthetic_248 = hmm_synthetic_248;
  ObsSeqs seqs;
  seqs.data = seqs_data;
  seqs.lens = seqs_lens;
  seqs.maxlen = seqs_maxlen;
  seqs.num_instances = seqs_num_instances;

  float neginf = -1.0 / 0.0;
  forward_init<1024><<<seqs.num_instances, 1024>>>(hmm, seqs, alpha1);
  for (int64_t t = 1; t < seqs.maxlen; t++) {
    forward_step<1024><<<seqs.num_instances, 1024>>>(hmm, seqs, alpha1, alpha2, t, neginf);
  }
  forward_lse<512><<<seqs.num_instances, 512>>>(hmm, seqs, alpha1, alpha2, neginf, result);
}

extern "C"
void forward_merged(
    float hmm_gamma, float *hmm_trans1, float *hmm_trans2,
    float *hmm_output_prob, float *hmm_initial_prob, float hmm_synthetic_248,
    int64_t hmm_num_states, uint8_t *seqs_data, int64_t *seqs_lens,
    int64_t seqs_maxlen, int64_t seqs_num_instances, float *result,
    float *alpha1, float *alpha2) {
  HMM hmm;
  hmm.gamma = hmm_gamma;
  hmm.trans1 = hmm_trans1;
  hmm.trans2 = hmm_trans2;
  hmm.output_prob = hmm_output_prob;
  hmm.initial_prob = hmm_initial_prob;
  hmm.synthetic_248 = hmm_synthetic_248;
  ObsSeqs seqs;
  seqs.data = seqs_data;
  seqs.lens = seqs_lens;
  seqs.maxlen = seqs_maxlen;
  seqs.num_instances = seqs_num_instances;

  float neginf = -1.0 / 0.0;
  forward_init<1024><<<seqs.num_instances, 1024>>>(hmm, seqs, alpha1);
  forward_steps<1024><<<seqs.num_instances, 1024>>>(hmm, seqs, alpha1, alpha2, neginf);
  forward_lse<512><<<seqs.num_instances, 512>>>(hmm, seqs, alpha1, alpha2, neginf, result);
}
